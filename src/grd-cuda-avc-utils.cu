/*
 * Copyright (C) 2021 Pascal Nowack
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License as
 * published by the Free Software Foundation; either version 2 of the
 * License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA
 * 02111-1307, USA.
 */

/*
 * Generate the PTX instructions with:
 * clang --cuda-gpu-arch=sm_30 -S src/grd-cuda-avc-utils.cu -o data/grd-cuda-avc-utils_30.ptx --no-cuda-version-check -O3 --cuda-device-only -Wall -Wextra
 *
 * or
 *
 * nvcc -arch=compute_30 -ptx grd-cuda-avc-utils.cu -o grd-cuda-avc-utils_30.ptx
 *
 * Note: This requires CUDA < 11, since the generation of Kepler capable
 * PTX code was removed from CUDA 11.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C"
{
  __device__ uint8_t
  rgb_to_y (uint8_t r,
            uint8_t g,
            uint8_t b)
  {
    return (54 * r + 183 * g + 18 * b) >> 8;
  }

  __device__ uint8_t
  rgb_to_u (uint8_t r,
            uint8_t g,
            uint8_t b)
  {
    return ((-29 * r - 99 * g + 128 * b) >> 8) + 128;
  }

  __device__ uint8_t
  rgb_to_v (uint8_t r,
            uint8_t g,
            uint8_t b)
  {
    return ((128 * r - 116 * g - 12 * b) >> 8) + 128;
  }

  __global__ void
  convert_2x2_bgrx_area_to_yuv420_nv12 (uint8_t  *dst_data,
                                        uint32_t *src_data,
                                        uint16_t  src_width,
                                        uint16_t  src_height,
                                        uint16_t  aligned_width,
                                        uint16_t  aligned_height,
                                        uint16_t  aligned_stride)
  {
    uint8_t *dst_y0, *dst_y1, *dst_y2, *dst_y3, *dst_u, *dst_v;
    uint32_t *src_u32;
    uint16_t s0, s1, s2, s3;
    uint32_t bgrx;
    int32_t r_a, g_a, b_a;
    uint8_t r, g, b;
    uint16_t x_1x1, y_1x1;
    uint16_t x_2x2, y_2x2;

    x_2x2 = blockIdx.x * blockDim.x + threadIdx.x;
    y_2x2 = blockIdx.y * blockDim.y + threadIdx.y;

    if (x_2x2 >= aligned_width >> 1 || y_2x2 >= aligned_height >> 1)
      return;

    /*
     *  -------------
     *  | d_0 | d_1 |
     *  -------------
     *  | d_2 | d_3 |
     *  -------------
     */
    s0 = 0;
    s1 = 1;
    s2 = src_width;
    s3 = src_width + 1;

    x_1x1 = x_2x2 << 1;
    y_1x1 = y_2x2 << 1;
    src_u32 = src_data + y_1x1 * src_width + x_1x1;

    dst_y0 = dst_data + y_1x1 * aligned_stride + x_1x1;
    dst_y1 = dst_y0 + 1;
    dst_y2 = dst_data + (y_1x1 + 1) * aligned_stride + x_1x1;
    dst_y3 = dst_y2 + 1;
    dst_u = dst_data + aligned_height * aligned_stride +
            y_2x2 * aligned_stride + x_1x1;
    dst_v = dst_u + 1;

    /* d_0 */
    if (x_1x1 < src_width && y_1x1 < src_height)
      {
        bgrx = src_u32[s0];

        b_a = b = *(((uint8_t *) &bgrx) + 0);
        g_a = g = *(((uint8_t *) &bgrx) + 1);
        r_a = r = *(((uint8_t *) &bgrx) + 2);
        *dst_y0 = rgb_to_y (r, g, b);
      }
    else
      {
        b_a = b = 0;
        g_a = g = 0;
        r_a = r = 0;
        *dst_y0 = 0;
      }

    if (x_1x1 + 1 < src_width && y_1x1 < src_height)
      {
        bgrx = src_u32[s1];

        /* d_1 */
        b_a += b = *(((uint8_t *) &bgrx) + 0);
        g_a += g = *(((uint8_t *) &bgrx) + 1);
        r_a += r = *(((uint8_t *) &bgrx) + 2);
        *dst_y1 = rgb_to_y (r, g, b);
      }
    else
      {
        *dst_y1 = 0;
      }

    if (x_1x1 < src_width && y_1x1 + 1 < src_height)
      {
        bgrx = src_u32[s2];

        /* d_2 */
        b_a += b = *(((uint8_t *) &bgrx) + 0);
        g_a += g = *(((uint8_t *) &bgrx) + 1);
        r_a += r = *(((uint8_t *) &bgrx) + 2);
        *dst_y2 = rgb_to_y (r, g, b);

        if (x_1x1 + 1 < src_width)
          {
            bgrx = src_u32[s3];

            /* d_3 */
            b_a += b = *(((uint8_t *) &bgrx) + 0);
            g_a += g = *(((uint8_t *) &bgrx) + 1);
            r_a += r = *(((uint8_t *) &bgrx) + 2);
            *dst_y3 = rgb_to_y (r, g, b);
          }
        else
          {
            *dst_y3 = 0;
          }
      }
    else
      {
        *dst_y2 = 0;
        *dst_y3 = 0;
      }

    b_a >>= 2;
    g_a >>= 2;
    r_a >>= 2;
    *dst_u = rgb_to_u (r_a, g_a, b_a);
    *dst_v = rgb_to_v (r_a, g_a, b_a);
  }
}
