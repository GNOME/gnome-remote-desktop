/*
 * Copyright (C) 2021 Pascal Nowack
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License as
 * published by the Free Software Foundation; either version 2 of the
 * License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA
 * 02111-1307, USA.
 */

/*
 * Generate the PTX instructions with:
 * nvcc -arch=compute_30 -ptx grd-cuda-avc-utils.cu -o grd-cuda-avc-utils_30.ptx
 *
 * Note: This requires CUDA < 11, since the generation of Kepler capable
 * PTX code was removed from CUDA 11.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C"
{
  __device__ uint16_t
  nv12_get_interlaced_y_1x1 (uint16_t y_1x1,
                             uint16_t aligned_height)
  {
    if (y_1x1 < aligned_height >> 1)
      return y_1x1 << 1;
    return (y_1x1 << 1) - aligned_height + 1;
  }

  __device__ uint16_t
  nv12_get_interlaced_y_2x2 (uint16_t y_2x2,
                             uint16_t aligned_height)
  {
    if (y_2x2 < aligned_height >> 2)
      return y_2x2 << 1;
    return (y_2x2 << 1) - (aligned_height >> 1) + 1;
  }

  __device__ uint8_t
  rgb_to_y (uint8_t r,
            uint8_t g,
            uint8_t b)
  {
    return (54 * r + 183 * g + 18 * b) >> 8;
  }

  __device__ uint8_t
  rgb_to_u (uint8_t r,
            uint8_t g,
            uint8_t b)
  {
    return ((-29 * r - 99 * g + 128 * b) >> 8) + 128;
  }

  __device__ uint8_t
  rgb_to_v (uint8_t r,
            uint8_t g,
            uint8_t b)
  {
    return ((128 * r - 116 * g - 12 * b) >> 8) + 128;
  }

  __global__ void
  convert_2x2_bgrx_area_to_yuv420_nv12 (uint8_t  *dst_data,
                                        uint8_t  *src_data,
                                        uint16_t  src_width,
                                        uint16_t  src_height,
                                        uint16_t  src_stride,
                                        uint16_t  aligned_width,
                                        uint16_t  aligned_height,
                                        uint16_t  aligned_stride)
  {
    uint8_t *src, *dst_y0, *dst_y1, *dst_y2, *dst_y3, *dst_u, *dst_v;
    uint16_t s0, s1, s2, s3;
    int32_t r_a, g_a, b_a;
    uint8_t r, g, b;
    uint16_t x_1x1, y_1x1;
    uint16_t x_2x2, y_2x2;

    x_2x2 = blockIdx.x * blockDim.x + threadIdx.x;
    y_2x2 = blockIdx.y * blockDim.y + threadIdx.y;

    if (x_2x2 >= aligned_width >> 1 || y_2x2 >= aligned_height >> 1)
      return;

    /*
     *  -------------
     *  | d_0 | d_1 |
     *  -------------
     *  | d_2 | d_3 |
     *  -------------
     */
    s0 = 0;
    s1 = 4;
    s2 = src_stride;
    s3 = src_stride + 4;
    /*
     * Technically, the correct positions for the Y data in the resulting NV12
     * image would be the following:
     *
     * d0 = 0;
     * d1 = 1;
     * d2 = aligned_stride;
     * d3 = aligned_stride + 1;
     *
     * However, since MBAFF is used as frame field mode, NVENC requires the input
     * frame to be interlaced.
     * If the frame is not interlaced, then even lines end up in the position
     * y / 2, instead of y and odd lines end up in the position y / 2 +
     * aligned_height / 2, instead of y.
     * So, calculate the interlaced y position via a dedicated function, which
     * ensures that the lines in the input frame end up in the resulting frame to
     * be at the correct position.
     * Doing this now in the kernel here, instead of after the BGRX -> YUV420
     * conversion, saves a huge amount of time, since each thread only has a
     * super tiny overhead to perform this action, while a normal
     * device-to-device copy operation can take at least several milliseconds.
     */

    x_1x1 = x_2x2 << 1;
    y_1x1 = y_2x2 << 1;
    src = src_data + y_1x1 * src_stride + (x_1x1 << 2);

    dst_y0 = dst_data +
             nv12_get_interlaced_y_1x1 (y_1x1, aligned_height) * aligned_stride +
             x_1x1;
    dst_y1 = dst_y0 + 1;
    dst_y2 = dst_data +
             nv12_get_interlaced_y_1x1 (y_1x1 + 1, aligned_height) * aligned_stride +
             x_1x1;
    dst_y3 = dst_y2 + 1;
    dst_u = dst_data + aligned_height * aligned_stride +
            nv12_get_interlaced_y_2x2 (y_2x2, aligned_height) * aligned_stride +
            x_1x1;
    dst_v = dst_u + 1;

    /* d_0 */
    if (x_1x1 < src_width && y_1x1 < src_height)
      {
        b_a = b = src[s0 + 0];
        g_a = g = src[s0 + 1];
        r_a = r = src[s0 + 2];
        *dst_y0 = rgb_to_y (r, g, b);
      }
    else
      {
        b_a = b = 0;
        g_a = g = 0;
        r_a = r = 0;
        *dst_y0 = 0;
      }

    if (x_1x1 + 1 < src_width && y_1x1 < src_height)
      {
        /* d_1 */
        b_a += b = src[s1 + 0];
        g_a += g = src[s1 + 1];
        r_a += r = src[s1 + 2];
        *dst_y1 = rgb_to_y (r, g, b);
      }
    else
      {
        *dst_y1 = 0;
      }

    if (x_1x1 < src_width && y_1x1 + 1 < src_height)
      {
        /* d_2 */
        b_a += b = src[s2 + 0];
        g_a += g = src[s2 + 1];
        r_a += r = src[s2 + 2];
        *dst_y2 = rgb_to_y (r, g, b);

        if (x_1x1 + 1 < src_width)
          {
            /* d_3 */
            b_a += b = src[s3 + 0];
            g_a += g = src[s3 + 1];
            r_a += r = src[s3 + 2];
            *dst_y3 = rgb_to_y (r, g, b);
          }
        else
          {
            *dst_y3 = 0;
          }
      }
    else
      {
        *dst_y2 = 0;
        *dst_y3 = 0;
      }

    b_a >>= 2;
    g_a >>= 2;
    r_a >>= 2;
    *dst_u = rgb_to_u (r_a, g_a, b_a);
    *dst_v = rgb_to_v (r_a, g_a, b_a);
  }
}
