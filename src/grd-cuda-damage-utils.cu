/*
 * Copyright (C) 2021 Pascal Nowack
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License as
 * published by the Free Software Foundation; either version 2 of the
 * License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place - Suite 330, Boston, MA
 * 02111-1307, USA.
 */


#include <hip/hip_runtime.h>
#include <stdint.h>

extern "C"
{
  __global__ void
  check_damaged_pixel (uint8_t  *damage_array,
                       uint8_t  *region_is_damaged,
                       uint32_t *current_data,
                       uint32_t *previous_data,
                       uint32_t  damage_array_stride,
                       uint32_t  data_width,
                       uint32_t  data_height,
                       uint32_t  data_stride)
  {
    uint32_t data_pos;
    uint8_t damaged = 0;
    uint32_t x, y;

    x = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= data_width || y >= data_height)
      return;

    data_pos = y * data_stride + x;
    if (previous_data[data_pos] != current_data[data_pos])
      {
        damaged = 1;
        *region_is_damaged = 1;
      }

    damage_array[y * damage_array_stride + x] = damaged;
  }

  __global__ void
  combine_damage_array_cols (uint8_t  *damage_array,
                             uint32_t  damage_array_width,
                             uint32_t  damage_array_height,
                             uint32_t  damage_array_stride,
                             uint32_t  combine_shift)
  {
    uint32_t data_pos;
    uint32_t neighbour_offset;
    uint32_t x, y;
    uint32_t sx;

    sx = blockIdx.x * blockDim.x + threadIdx.x;
    y = blockIdx.y * blockDim.y + threadIdx.y;

    x = sx << combine_shift + 1;

    if (x >= damage_array_width || y >= damage_array_height)
      return;

    neighbour_offset = 1 << combine_shift;
    if (x + neighbour_offset >= damage_array_width)
      return;

    data_pos = y * damage_array_stride + x;
    if (damage_array[data_pos + neighbour_offset])
      damage_array[data_pos] = 1;
  }

  __global__ void
  combine_damage_array_rows (uint8_t  *damage_array,
                             uint32_t  damage_array_width,
                             uint32_t  damage_array_height,
                             uint32_t  damage_array_stride,
                             uint32_t  combine_shift)
  {
    uint32_t data_pos;
    uint32_t neighbour_offset;
    uint32_t x, y;
    uint32_t sy;

    x = blockIdx.x * blockDim.x + threadIdx.x;
    sy = blockIdx.y * blockDim.y + threadIdx.y;

    y = sy << combine_shift + 1;

    if (x >= damage_array_width || y >= damage_array_height)
      return;

    neighbour_offset = 1 << combine_shift;
    if (y + neighbour_offset >= damage_array_height)
      return;

    data_pos = y * damage_array_stride + x;
    if (damage_array[data_pos + neighbour_offset * damage_array_stride])
      damage_array[data_pos] = 1;
  }

  __global__ void
  simplify_damage_array (uint8_t  *dst_damage_array,
                         uint8_t  *src_damage_array,
                         uint32_t  dst_damage_array_stride,
                         uint32_t  src_damage_array_width,
                         uint32_t  src_damage_array_height,
                         uint32_t  src_damage_array_stride)
  {
    uint32_t src_data_pos, dst_data_pos;
    uint32_t sx, sy;
    uint32_t x, y;

    sx = blockIdx.x * blockDim.x + threadIdx.x;
    sy = blockIdx.y * blockDim.y + threadIdx.y;

    x = sx << 6;
    y = sy << 6;

    if (x >= src_damage_array_width || y >= src_damage_array_height)
      return;

    src_data_pos = y * src_damage_array_stride + x;
    dst_data_pos = sy * dst_damage_array_stride + sx;

    dst_damage_array[dst_data_pos] = src_damage_array[src_data_pos];
  }
}
